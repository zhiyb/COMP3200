#include <stdint.h>
#include <string.h>
#include <opencv2/opencv.hpp>
#include <opencv2/gpu/gpu.hpp>

void bayer10toRGB(unsigned int width, unsigned int height, void *input, void *output)
{
	const unsigned int nInput = width * height * 2;
	const unsigned int nOutput = width * height * 3;
	// Allocate device memory
	void *dInput = 0, *dOutput = 0;
	hipMalloc(&dInput, nInput);
	hipMalloc(&dOutput, nOutput);

	// Copy input data to device
	hipMemcpy(dInput, input, nInput, hipMemcpyHostToDevice);

	// Execute the kernel
	//gpuBayer10toRGB<<<?, dim3()>>>(dInput, dOutput);

	// Copy output data from device
	hipMemcpy(output, dOutput, nOutput, hipMemcpyDeviceToHost);
	//memcpy(output, input, width * height * 3);

	hipFree(dInput);
	hipFree(dOutput);
}
